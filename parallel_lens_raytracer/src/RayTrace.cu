#include "hip/hip_runtime.h"
#pragma once
#include "Camera.h"
#include "Intersection.h"
#include "Parameters.h"
#include "Phases.h"
#include "Ray.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hiprand/hiprand_kernel.h"
#include ""
// clang-format off
#include "glad/gl.h"
#include "cuda_gl_interop.h"
// clang-format on
#include "vec3.h"

#include <float.h>
#include <hip/hip_vector_types.h>

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

struct Texture
{
  GLuint textureId;
  struct hipGraphicsResource* cudaResource;
  hipSurfaceObject_t viewCudaSurfaceObject;
};

template<typename T>
void check(T err, const char* const func, const char* const file, const int line)
{
  if (err != hipSuccess)
  {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

// __device__ static float Radians(const float degrees)
// {
//   return M_PI * degrees / 180.f;
// }

__device__ static float fresnelAR(float theta0, float lambda, float d, float n0, float n1, float n2)
{
  // Apply Snell's law to get the other angles
  float theta1 = asin(sin(theta0) * n0 / n1);
  float theta2 = asin(sin(theta0) * n0 / n2);

  float rs01 = -sin(theta0 - theta1) / sin(theta0 + theta1);
  float rp01 = tan(theta0 - theta1) / tan(theta0 + theta1);
  float ts01 = 2.0 * sin(theta1) * cos(theta0) / sin(theta0 + theta1);
  float tp01 = ts01 * cos(theta0 - theta1);

  float rs12 = -sin(theta1 - theta2) / sin(theta1 + theta2);
  float rp12 = tan(theta1 - theta2) / tan(theta1 + theta2);

  float ris = ts01 * ts01 * rs12;
  float rip = tp01 * tp01 * rp12;

  float dy = d * n1;
  float dx = tan(theta1) * dy;
  float delay = sqrt(dx * dx + dy * dy);
  float relPhase = 4.0 * M_PI / lambda * (delay - dx * sin(theta0));

  float out_s2 = rs01 * rs01 + ris * ris + 2.0f * rs01 * ris * cos(relPhase);
  float out_p2 = rp01 * rp01 + rip * rip + 2.0f * rp01 * rip * cos(relPhase);

  return (out_s2 + out_p2) * 0.5f;
}

__device__ Vec3 Reflect(const Vec3& incident, const Vec3& normal)
{
  return incident - 2 * dot(incident, normal) * normal;
}

__device__ Vec3 Refract(const Vec3& incident, const Vec3& normal, float eta)
{
  float k = 1.0f - eta * eta * (1.0f - dot(normal, incident) * dot(normal, incident));
  if (k < 0.0)
    return {0.f, 0.f, 0.f};
  return eta * incident - (eta * dot(normal, incident) + sqrtf(k)) * normal;
}

__host__ __device__ uint32_t CountNumberOfInterfacesInvolved(const Camera& camera, const Ghost& ghost)
{
  uint32_t counterInterfaces{0U};
  for (uint32_t i = 0; i < ghost.lensIndexOne; i++)
    counterInterfaces++;
  // Phase 1
  for (uint32_t i = ghost.lensIndexOne; i > ghost.lensIndexTwo; i--)
    counterInterfaces++;
  // Phase 2
  for (uint32_t i = ghost.lensIndexTwo; i < camera.GetNumberOfInterfaces(); i++)
    counterInterfaces++;
  return counterInterfaces;
}

__host__ void CreateTexture(const Parameters& parameters,
                            GLuint* textureId,
                            struct hipGraphicsResource** resource,
                            unsigned int resFlags,
                            const std::vector<uint8_t>& pixels)
{
  glGenTextures(1, textureId);
  glBindTexture(GL_TEXTURE_2D, *textureId);
  {
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexImage2D(GL_TEXTURE_2D,
                 0,
                 GL_RGBA,
                 parameters.samplesInX,
                 parameters.samplesInY,
                 0,
                 GL_RGBA,
                 GL_UNSIGNED_BYTE,
                 pixels.data());
  }
  // glBindTexture(GL_TEXTURE_2D, 0);

  checkCudaErrors(hipGraphicsGLRegisterImage(resource, *textureId, GL_TEXTURE_2D, resFlags));
}

__global__ void setupKernel(hiprandState* state, int w, int h, int numGhosts)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  int ghostIndex = blockDim.z * blockIdx.z + threadIdx.z;
  if (x >= w || y >= h || ghostIndex >= numGhosts)
    return;
  auto index = (ghostIndex * w * h) + (y * w + x);
  // Each thread gets same seed, different suquence, no offset
  hiprand_init(1111, index, 0, &state[index]);
}

__global__ void
  FindMax(const Parameters& parameters, const int numberOfGhosts, float3* sensorIntersections, float2* maxValue)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  int ghostIndex = blockDim.z * blockIdx.z + threadIdx.z;
  const auto w = parameters.samplesInX;
  const auto h = parameters.samplesInY;
  if (x >= w || y >= h || ghostIndex >= numberOfGhosts)
    return;
  int lambdaFor = (parameters.spectral) ? 3 : 1;
  for (int l = 0; l < lambdaFor; l++)
  {
    auto index = (3 * ghostIndex + l) * (w * h) + (y * w + x);
    const float3 sensorVal = sensorIntersections[index];
    if (auto v = make_float2(sensorVal.x, sensorVal.y);
        v.x > maxValue[ghostIndex].x && v.y > maxValue[ghostIndex].y)
    {
      maxValue[ghostIndex] = v;
    }
  }
}
__global__ void
  FindMin(const Parameters& parameters, const int numberOfGhosts, float3* sensorIntersections, float2* minValue)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  int ghostIndex = blockDim.z * blockIdx.z + threadIdx.z;
  const auto w = parameters.samplesInX;
  const auto h = parameters.samplesInY;
  if (x >= w || y >= h || ghostIndex >= numberOfGhosts)
    return;
  int lambdaFor = (parameters.spectral) ? 3 : 1;
  for (int l = 0; l < lambdaFor; l++)
  {
    auto index = (3 * ghostIndex + l) * (w * h) + (y * w + x);
    const float3 sensorVal = sensorIntersections[index];
    if (auto v = make_float2(sensorVal.x, sensorVal.y);
        v.x > minValue[ghostIndex].x && v.y > minValue[ghostIndex].y)
    {
      minValue[ghostIndex] = v;
    }
  }
}

__global__ void CalculateTextures(const Parameters& parameters,
                                  const Ghost* ghosts,
                                  const int ghostIndex,
                                  float3* sensorIntersections,
                                  hipSurfaceObject_t viewCudaSurfaceObject)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  const auto w = parameters.samplesInX;
  const auto h = parameters.samplesInY;
  if (x >= w || y >= h)
    return;

  const Camera& camera = parameters.camera;
  const int lambdaFor = (parameters.spectral) ? 3 : 1;
  float delta_u = camera.filmWidth / (float)w;
  float delta_v = camera.filmHeight / (float)h;
  Vec3 txtColor;
  // float scale = (1 / (float)lambdaFor);
  uchar4 prevColor;
  int gridX{0}, gridY{0};
  for (int l = 0; l < lambdaFor; l++)
  {
    auto index = (3 * ghostIndex + l) * (w * h) + (y * w + x);
    gridX = (int)floor((sensorIntersections[index].x + camera.filmWidth / 2.f) / delta_u);
    gridY = (int)floor((sensorIntersections[index].y + camera.filmHeight / 2.f) / delta_v);
    surf2Dread(&prevColor, viewCudaSurfaceObject, gridX * sizeof(uchar4), gridY);
    Vec3 lastColor{prevColor};
    Vec3 lightColor = (parameters.spectral) ? lambda2RGB(parameters.light.lambda[l], 1.f) : parameters.light.color;
    Vec3 color = {sensorIntersections[index].z * lightColor.x(),
                  sensorIntersections[index].z * lightColor.y(),
                  sensorIntersections[index].z * lightColor.z()};
    if (lastColor.near_zero())
    {
      txtColor = color;
    }
    else
    {
      txtColor = lastColor + color;
      txtColor *= 0.5f;
    }
  }
  uchar3 color = txtColor.touchar3();
  uchar4 c4 = make_uchar4(color.x, color.y, color.z, 255);
  surf2Dwrite(c4, viewCudaSurfaceObject, gridX * sizeof(uchar4), gridY);
}

/**
 * @brief Kernel that traces a light sampled from its position through all camera lenses interfaces up to the last one
 * 
 * @param camera Camera
 * @param light Light to be sampled
 * @param ghostEnumeration Vector of ghost that produces the flare
 * @param numberOfGhost Number of ghosts
 * @param numSamplesX number of samples in X direction the width of the light is divided
 * @param numSamplesY number of samples in Y direction the height of the light is divided
 * @param rayOut Ray that exits the last element of the interface
 * @param apertureIntersection Where the ray intersects the camera aperture plane
 * @param lensIntersection Where the ray intersects the camera interfaces
 */
__global__ void ParallelRayTrace(const Parameters parameters,
                                 const Ghost* ghosts,
                                 const int numberOfGhosts,
                                 hiprandState* rndStates,
                                 float3* pupilIntersections,
                                 float3* sensorIntersections,
                                 float2* apertureIntersection)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  int ghostIndex = blockDim.z * blockIdx.z + threadIdx.z;
  const auto w = parameters.samplesInX;
  const auto h = parameters.samplesInY;
  if (x >= w || y >= h || ghostIndex >= numberOfGhosts)
    return;

  const Camera& camera = parameters.camera;
  float distanceX = camera.InterfaceAt(1).apertureDiameter / (float)parameters.division;
  float distanceY = camera.InterfaceAt(1).apertureDiameter / (float)parameters.division;
  Ghost ghost{-1, -1};

  int indexInterface{0};
  Phase phase{Phase::Zero};

  const Vec3 horizontal{(float)(parameters.width)};
  const Vec3 vertical{0.f, -(float)(parameters.height)};
  const float delta_u = distanceX / (float)w;
  const float delta_v = distanceY / (float)h;
  const Vec3 lightPosition = camera.InterfaceAt(0).position + parameters.light.position;
  // const Vec3 gridUpperLeft =
  //   camera.InterfaceAt(0).position + parameters.light.position - 0.5f * (horizontal + vertical);

  hiprandState rndState = rndStates[(ghostIndex * w * h) + (y * w + x)];

  int lambdaFor = (parameters.spectral) ? 3 : 1;
  for (int l = 0; l < lambdaFor; l++)
  {
    float3 pupilPosition = make_float3(0.f, 0.f, 0.f);
    if (ghostIndex != numberOfGhosts - 1)
    {
      ghost = ghosts[ghostIndex];
      distanceX = ghost.bounds[l].x * 2.f / (float)parameters.division;
      distanceY = ghost.bounds[l].y * 2.f / (float)parameters.division;
    }
    auto index = (3 * ghostIndex + l) * (w * h) + (y * w + x);

    float delta_x = -0.5f + hiprand_uniform(&rndState);
    float delta_y = -0.5f + hiprand_uniform(&rndState);

    // Build the ray in from the parameters
    Ray rayIn;
    // rayIn.origin = gridUpperLeft + (x + delta_x) * Vec3{delta_u} + (y + delta_y) * Vec3{0.f, -delta_v};
    rayIn.origin = lightPosition;
    float xCoordinate = -(distanceX / 2.f) + x * delta_u;
    float yCoordinate = (distanceY / 2.f) - y * delta_v;
    Vec3 direction = Vec3{xCoordinate, yCoordinate, camera.InterfaceAt(0).position.z()} - lightPosition;
    rayIn.direction = direction;
    rayIn.direction.make_unit_vector();

    // rayIn.direction = parameters.light.direction;
    // rayIn.direction = {delta_x * delta_u, delta_y * delta_v, rayIn.direction.z()};
    // rayIn.direction.make_unit_vector();
    rayIn.intensity = parameters.light.intensity;

    // Count the number of interfaces involved in that ghost
    uint32_t counterInterfaces = camera.GetNumberOfInterfaces();
    if (ghostIndex != numberOfGhosts - 1)
    {
      counterInterfaces = CountNumberOfInterfacesInvolved(camera, ghost);
    }
    Intersection intersection;
    for (int i = 0; i < counterInterfaces && indexInterface < camera.GetNumberOfInterfaces(); i++)
    {
      const LensInterface& interface = camera.InterfaceAt(indexInterface);
      const int iI{indexInterface};
      bool isSelected{false};
      switch (phase)
      {
        case Phase::Zero:
          isSelected = (indexInterface == ghost.lensIndexOne);
          if (isSelected)
          {
            phase = Phase::One;
            indexInterface--;
          }
          else
          {
            indexInterface++;
          }
          break;
        case Phase::One:
          isSelected = (indexInterface == ghost.lensIndexTwo);
          if (isSelected)
          {
            phase = Phase::Two;
            indexInterface++;
          }
          else
          {
            indexInterface--;
          }
          break;
        case Phase::Two:
          indexInterface++;
          break;
      }

      intersection = interface.GetIntersection(rayIn);
      if (intersection.hit && interface.type == LensInterface::Type::Aperture)
      {
        float2 uv = make_float2(intersection.position.x(), intersection.position.y());
        float radius = interface.apertureDiameter / 2.f;
        if (!camera.IntersectionWithAperture(uv, radius))
        {
          intersection.hit = false;
          break;
        }

        apertureIntersection[(ghostIndex * w * h) + y * w + x] = uv;
        rayIn.origin = intersection.position;
        continue;
      }
      else if (!intersection.hit)
      {
        break;
      }

      if (i == 0)
      {
        pupilPosition = make_float3(intersection.position.x(), intersection.position.y(), 0.f);
      }

      // tmpInterface is next one
      // interface is the current
      // Ray in z < 0 travels to sensor: prevInterface is iI-1
      // Ray in z > 0 travels to front camera: prevInterface is iI+1
      const int prevInterfaceIndex = (rayIn.direction.z() < 0.f) ? iI - 1 : iI + 1;
      float n0 = 1.f;
      if (prevInterfaceIndex >= 0 && prevInterfaceIndex < camera.GetNumberOfInterfaces())
      {
        const auto prevInterface = camera.InterfaceAt(prevInterfaceIndex);
        n0 = (parameters.spectral) ? prevInterface.ComputeIOR(parameters.light.lambda[l]) : prevInterface.ior;
      }

      float n2 = 1.f;
      n2 = (parameters.spectral) ? interface.ComputeIOR(parameters.light.lambda[l]) : interface.ior;

      rayIn.direction = unit_vector(intersection.position - rayIn.origin);
      if (intersection.inverted)
        rayIn.direction *= -1.f;

      if (isSelected)
      {
        rayIn.direction = Reflect(rayIn.direction, intersection.normal);
        float n1 = max(sqrt(n0 * n2), interface.coatingIor);
        float d1 = interface.coatingLambda / 4.0f / n1;
        float R = fresnelAR(intersection.theta, parameters.light.lambda[l], d1, n0, n1, n2);
        rayIn.intensity *= R;
      }
      else
      {
        rayIn.direction = Refract(rayIn.direction, intersection.normal, n0 / n2);
        if (rayIn.direction.near_zero())
        {
          rayIn.intensity = 0.f;
          break;
        }
      }
      rayIn.origin = intersection.position;
    }

    if (intersection.hit && indexInterface == camera.GetNumberOfInterfaces())
    {
      float2 uv = make_float2(intersection.position.x(), intersection.position.y());
      if (std::abs(uv.x) <= (float)(camera.filmWidth / 2.f) && std::abs(uv.y) <= (float)(camera.filmHeight / 2.f))
      {
        // rayIn.intensity = clamp(rayIn.intensity);
        sensorIntersections[index] =
          make_float3(intersection.position.x(), intersection.position.y(), rayIn.intensity);
        pupilIntersections[index] = pupilPosition;
      }
    }
  }
}

void RayTrace(const Parameters& parameters,
              std::vector<float3>& sensorIntersections,
              std::vector<float2> intersectionsWithAperture,
              std::vector<unsigned int>& texturesId)
{
  const auto& ghosts = parameters.camera.GetGhosts();
  uint32_t numGhosts = static_cast<uint32_t>(ghosts.size()) + 1; // Last one reserved for render the light
  uint32_t numInterfaces = static_cast<uint32_t>(parameters.camera.interfaces.size());

  // 3 dimensions (x,y,ghosts)
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  unsigned int threads_per_block = powf(prop.maxThreadsPerBlock, 1 / 3.f);
  dim3 blockSize(threads_per_block - 1, threads_per_block - 1, threads_per_block - 1);
  dim3 gridSize(ceil(parameters.samplesInX / (float)blockSize.x),
                ceil(parameters.samplesInY / (float)blockSize.y),
                ceil(numGhosts / (float)blockSize.z));

  uint32_t numRays = parameters.samplesInX * parameters.samplesInY * 3;

  // Reserve memory in device
  Parameters params = parameters;
  LensInterface* d_interfaces;
  hipMalloc((void**)&d_interfaces, numInterfaces * sizeof(LensInterface));
  checkCudaErrors(hipMemcpy(d_interfaces,
                             parameters.camera.interfaces.data(),
                             numInterfaces * sizeof(LensInterface),
                             hipMemcpyHostToDevice));
  params.camera.pInterfaces = d_interfaces;
  Ghost* d_ghosts;
  hipMalloc((void**)&d_ghosts, numGhosts * sizeof(Ghost));
  checkCudaErrors(hipMemcpy(d_ghosts, ghosts.data(), numGhosts * sizeof(Ghost), hipMemcpyHostToDevice));
  float3* h_tmp = (float3*)malloc(numRays * numGhosts * sizeof(float3));
  float2* h_tmp_1 = (float2*)malloc(numRays * numGhosts * sizeof(float2));
  float3* h_tmp_2 = (float3*)malloc(numRays * numGhosts * sizeof(float3));
  for (int i = 0; i < numRays * numGhosts; i++)
  {
    h_tmp[i] = make_float3(0.f, 0.f, 0.f);
    h_tmp_1[i] = make_float2(0.f, 0.f);
    h_tmp_2[i] = make_float3(0.f, 0.f, 0.f);
  }
  float3* d_pupilIntersections;
  float3* d_sensorIntersections;
  float2* d_apertureIntersections;
  hipMalloc((void**)&d_sensorIntersections, numRays * numGhosts * sizeof(float3));
  hipMalloc((void**)&d_apertureIntersections, numRays * numGhosts * sizeof(float2));
  hipMalloc((void**)&d_pupilIntersections, numRays * numGhosts * sizeof(float3));
  hipMemcpy(d_sensorIntersections, h_tmp, numRays * numGhosts * sizeof(float3), hipMemcpyHostToDevice);
  hipMemcpy(d_apertureIntersections, h_tmp_1, numRays * numGhosts * sizeof(float2), hipMemcpyHostToDevice);
  hipMemcpy(d_pupilIntersections, h_tmp_2, numRays * numGhosts * sizeof(float3), hipMemcpyHostToDevice);

  // Randon numbers
  hiprandState* d_states;
  checkCudaErrors(hipMalloc((void**)&d_states,
                             sizeof(hiprandState) * parameters.samplesInX * parameters.samplesInY * numGhosts));
  setupKernel<<<gridSize, blockSize>>>(d_states, parameters.samplesInX, parameters.samplesInY, numGhosts);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  ParallelRayTrace<<<gridSize, blockSize>>>(params,
                                            d_ghosts,
                                            numGhosts,
                                            d_states,
                                            d_pupilIntersections,
                                            d_sensorIntersections,
                                            d_apertureIntersections);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  float2* pupilMax = (float2*)malloc(numGhosts * sizeof(float2));
  float2* sensorMax = (float2*)malloc(numGhosts * sizeof(float2));
  float2* pupilMin = (float2*)malloc(numGhosts * sizeof(float2));
  float2* sensorMin = (float2*)malloc(numGhosts * sizeof(float2));
  for (int i = 0; i < numGhosts; i++)
  {
    pupilMax[i] = make_float2(FLT_MIN, FLT_MIN);
    sensorMax[i] = make_float2(FLT_MIN, FLT_MIN);
    pupilMin[i] = make_float2(FLT_MAX, FLT_MAX);
    sensorMin[i] = make_float2(FLT_MAX, FLT_MAX);
  }
  float2* d_pupilMax;
  float2* d_sensorMax;
  hipMalloc((void**)&d_pupilMax, numGhosts * sizeof(float2));
  hipMalloc((void**)&d_sensorMax, numGhosts * sizeof(float2));
  hipMemcpy(d_pupilMax, pupilMax, numGhosts * sizeof(float2), hipMemcpyHostToDevice);
  hipMemcpy(d_sensorMax, sensorMax, numGhosts * sizeof(float2), hipMemcpyHostToDevice);

  float2* d_pupilMin;
  float2* d_sensorMin;
  hipMalloc((void**)&d_pupilMin, numGhosts * sizeof(float2));
  hipMalloc((void**)&d_sensorMin, numGhosts * sizeof(float2));
  hipMemcpy(d_pupilMin, pupilMin, numGhosts * sizeof(float2), hipMemcpyHostToDevice);
  hipMemcpy(d_sensorMin, sensorMin, numGhosts * sizeof(float2), hipMemcpyHostToDevice);

  FindMax<<<gridSize, blockSize>>>(params, numGhosts, d_sensorIntersections, d_sensorMax);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  FindMax<<<gridSize, blockSize>>>(params, numGhosts, d_pupilIntersections, d_pupilMax);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  FindMin<<<gridSize, blockSize>>>(params, numGhosts, d_sensorIntersections, d_sensorMin);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  FindMin<<<gridSize, blockSize>>>(params, numGhosts, d_pupilIntersections, d_pupilMin);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  hipMemcpy(sensorMin, d_sensorMin, numGhosts * sizeof(float2), hipMemcpyDeviceToHost);
  hipMemcpy(pupilMin, d_pupilMin, numGhosts * sizeof(float2), hipMemcpyDeviceToHost);
  hipMemcpy(pupilMax, d_pupilMax, numGhosts * sizeof(float2), hipMemcpyDeviceToHost);
  hipMemcpy(sensorMax, d_sensorMax, numGhosts * sizeof(float2), hipMemcpyDeviceToHost);

  ////////////////////////////////////////////////////////////////////////////////
  // cuda open gl interop
  ////////////////////////////////////////////////////////////////////////////////

  // std::vector<uint8_t> pixels;
  // pixels.insert(pixels.begin(), parameters.samplesInX * parameters.samplesInY * 4, 0U);
  // for (int i = 0; i < numGhosts; i++)
  // {
  //   texturesId.push_back(0);
  //   auto& textureId = texturesId.back();
  //   struct hipGraphicsResource* cudaResource;
  //   CreateTexture(parameters, &textureId, &cudaResource, cudaGraphicsMapFlagsWriteDiscard, pixels);
  //   checkCudaErrors(hipGraphicsMapResources(1, &cudaResource, 0));
  //   {
  //     hipArray_t viewCudaArray;
  //     hipGraphicsSubResourceGetMappedArray(&viewCudaArray, cudaResource, 0, 0);
  //     hipResourceDesc viewCudaArrayResourceDesc;
  //     {
  //       viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
  //       viewCudaArrayResourceDesc.res.array.array = viewCudaArray;
  //     }

  //     hipSurfaceObject_t viewCudaSurfaceObject;
  //     hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc);
  //     {
  //       // 2 dimensions (x,y)
  //       threads_per_block = powf(prop.maxThreadsPerBlock, 1 / 2.f);
  //       blockSize = dim3(threads_per_block - 1, threads_per_block - 1);
  //       gridSize =
  //         dim3(ceil(parameters.samplesInX / (float)blockSize.x), ceil(parameters.samplesInY / (float)blockSize.y));
  //       CalculateTextures<<<gridSize, blockSize>>>(params,
  //                                                  d_ghosts,
  //                                                  i,
  //                                                  d_sensorIntersections,
  //                                                  viewCudaSurfaceObject);
  //       checkCudaErrors(hipGetLastError());
  //       checkCudaErrors(hipDeviceSynchronize());
  //     }
  //     hipDestroySurfaceObject(viewCudaSurfaceObject);
  //   }
  //   hipGraphicsUnmapResources(1, &cudaResource);
  // }

  hipMemcpy(h_tmp, d_sensorIntersections, numRays * numGhosts * sizeof(float3), hipMemcpyDeviceToHost);
  sensorIntersections.assign(h_tmp, h_tmp + (numRays * numGhosts));

  hipMemcpy(h_tmp_1, d_apertureIntersections, numRays * numGhosts * sizeof(float2), hipMemcpyDeviceToHost);
  intersectionsWithAperture.assign(h_tmp_1, h_tmp_1 + (numRays * numGhosts));

  free(h_tmp);
  free(h_tmp_1);
  checkCudaErrors(hipFree(d_interfaces));
  checkCudaErrors(hipFree(d_ghosts));
  checkCudaErrors(hipFree(d_sensorIntersections));
  checkCudaErrors(hipFree(d_apertureIntersections));
  checkCudaErrors(hipFree(d_states));
}